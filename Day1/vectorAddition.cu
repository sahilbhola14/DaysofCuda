// Add two vector of size n
#include <hip/hip_runtime.h>
#include <iostream>

#define cudaCheck(ans) gpuAssert((ans), __FILE__, __LINE__);

void gpuAssert(hipError_t err, const char *file, int line){
    if (err != hipSuccess){
        printf("<Error>: %s in %s at line %d\n", hipGetErrorString(err), file, line);
        exit(EXIT_FAILURE);
    }
}


__global__ void vecAdd(int n, float *A, float *B, float *C){
    // Enter the kernel here
}

int main(){

    // Variable definitions
    int n = 10; // Size of vector
    int size = n * sizeof(float); // Size of the vector in bytes
    float *h_A, *h_B, *h_C;
    float *d_A, *d_B, *d_C;

    // Allocation of memory
    cudaCheck(hipMalloc((void**)&d_A, size));
    cudaCheck(hipMalloc((void**)&d_B, size));
    cudaCheck(hipMalloc((void**)&d_C, size));

    // Copy the data to the Device
    cudaCheck(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice));

    // Invoke the Kernel

    // Check the Kernel Launch
    
    // Copy the result back to host
    cudaCheck(hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost));

    // Free
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
