// Add two vector of size n
#include <hip/hip_runtime.h>
#include <iostream>

#define cudaCheck(ans) gpuAssert((ans), __FILE__, __LINE__);

inline void gpuAssert(hipError_t err, const char *file, int line){
    if (err != hipSuccess){
        printf("<Error>: %s in %s at line %d\n", hipGetErrorString(err), file, line);
        exit(EXIT_FAILURE);
    }
}


__global__ void vecAdd(int n, float *A, float *B, float *C){
    // Enter the kernel here
}

int main(){

    // Variable definitions
    int n = 10; // Size of vector
    int size = n * sizeof(float); // Size of the vector in bytes
    float *h_A = new float[n];
    float *h_B = new float[n];
    float *h_C = new float[n];
    float *d_A, *d_B, *d_C;

    // Allocation of host memory
    for (int i = 0; i<n; i++){
        h_A[i] = static_cast<float>(1.0);
        h_B[i] = static_cast<float>(1.0);
    }

    // Allocation of device memory
    cudaCheck(hipMalloc((void**)&d_A, size));
    cudaCheck(hipMalloc((void**)&d_B, size));
    cudaCheck(hipMalloc((void**)&d_C, size));

    // Copy the data to the Device
    cudaCheck(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice));

    // Invoke the Kernel

    // Check the Kernel Launch
    
    // Copy the result back to host
    cudaCheck(hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost));

    // Free
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
